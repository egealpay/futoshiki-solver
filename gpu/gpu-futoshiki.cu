#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include<sstream>
#include<fstream>

#define SIZE 5 //Matrix size
#define INPUTSIZE 2306451

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// Ege Alpay 19551

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ bool canUseInRow(int *matrix, int currentRowIndex, int value) {
    int startIndex = currentRowIndex * 5;
    for (int i = startIndex; i < startIndex + 5; i++) {
        if (matrix[i] == value)
            return false;
    }

    return true;
}

__device__ bool canUseInColumn(int *matrix, int currentColIndex, int value) {
    for (int i = 0; i < 5; i++) {
        if (matrix[currentColIndex + 5 * i] == value)
            return false;
    }

    return true;
}

__device__ bool
isSatisfyConstraints(int *matrix, int currentRow, int currentCol, int value, int *constraints, int numConstraints) {
    for (int i = 0; i < numConstraints; i++) {
        // -1 Since constraint index start from 1, not 0
        int startRow = constraints[4 * i] - 1;
        int startCol = constraints[4 * i + 1] - 1;
        int endRow = constraints[4 * i + 2] - 1;
        int endCol = constraints[4 * i + 3] - 1;


        if (currentRow == startRow && currentCol == startCol) {
            if (value < matrix[5 * endRow + endCol] && matrix[5 * endRow + endCol] != -1)
                return false;
        } else if (currentRow == endRow && currentCol == endCol) {
            if (value > matrix[5 * startRow + startCol] && matrix[5 * startRow + startCol] != -1)
                return false;
        }
    }

    return true;
}

__device__ bool isCurrentCellEmpty(int *matrix, int currentRow, int currentCol) {
    return matrix[5 * currentRow + currentCol] == -1;
}

__device__ bool
doesSatisfyRules(int *matrix, int currentRow, int currentCol, int value, int *constraints, int numConstraints) {
    return canUseInRow(matrix, currentRow, value) &&
           canUseInColumn(matrix, currentCol, value) &&
           isSatisfyConstraints(matrix, currentRow, currentCol, value, constraints, numConstraints) &&
           isCurrentCellEmpty(matrix, currentRow, currentCol);
}

__device__ bool canUseInRowCheck(int *matrix, int currentRowIndex, int currentColIndex, int value) {
    int startIndex = currentRowIndex * 5;
    for (int i = startIndex; i < startIndex + 5; i++) {
        if (5 * currentRowIndex + currentColIndex != i && matrix[i] == value)
            return false;
    }

    return true;
}

__device__ bool canUseInColumnCheck(int *matrix, int currentRowIndex, int currentColIndex, int value) {
    for (int row = 0; row < 5; row++) {
        if (currentRowIndex != row && matrix[currentColIndex + 5 * row] == value)
            return false;
    }

    return true;
}

__device__ bool isSolutionCorrect(int *matrix, int *constraints, int numConstraints) {
    for (int row = 0; row < 5; row++) {
        for (int col = 0; col < 5; col++) {
            int value = matrix[5 * row + col];
            // printf("Checking Row: %d, Col: %d, Value: %d \n", row, col, value);

            if (value < 1 || value > 5) {
                // printf("Invalid value: %d \n", value);
                return false;
            }

            if (!canUseInRowCheck(matrix, row, col, value)) {
                // printf("Already in use in ROW: %d \n", value);
                return false;
            }

            if (!canUseInColumnCheck(matrix, row, col, value)) {
                // printf("Already in use in COL: %d \n", value);
                return false;
            }

            if (!isSatisfyConstraints(matrix, row, col, value, constraints, numConstraints)) {
                // printf("Constraints Fails: %d \n", value);
                return false;
            }
        }
    }

    return true;
}

__device__ int *findEmptyCell(int *matrix) {
    int emptyCell[2]; // row, col

    for (int row = 0; row < 5; row++) {
        for (int col = 0; col < 5; col++) {
            if (matrix[row * 5 + col] == -1) {
                emptyCell[0] = row;
                emptyCell[1] = col;
                return emptyCell;
            }
        }
    }

    emptyCell[0] = -1;
    emptyCell[1] = -1;
    return emptyCell;
}

// Stack like implementation was used since recursion on GPU is not recommended
__device__ void solveSingleThread(int *matrix, int *constraints, int numConstraints) {
    bool solved = false;
    bool unroll = false;
    int stack[50];
    int stackIndex = 0;
    int values[25]; // 5*row + col

    // Every cells last taken value will be stored in this array
    for (int i = 0; i < 25; i++)
        values[i] = 1;

    while (!solved) {
        while (unroll) {
            int col = stack[--stackIndex];
            int row = stack[--stackIndex];

            matrix[row * 5 + col] = -1;

            if (values[5 * row + col] != 5) {
                unroll = false;
                values[5 * row + col] += 1;
            } else {
                values[5 * row + col] = 1;
            }
        }

        int *emptyCell = findEmptyCell(matrix);
        int row = emptyCell[0];
        int col = emptyCell[1];

        if (row == -1 && col == -1) {
            solved = true;
            break;
        }

        for (int value = values[5 * row + col]; value <= 5; value++) {
            if (doesSatisfyRules(matrix, row, col, value, constraints, numConstraints)) {
                matrix[row * 5 + col] = value;

                stack[stackIndex++] = row;
                stack[stackIndex++] = col;

                values[5 * row + col] = value;

                break;
            }

            if (value == 5)
                unroll = true;

            if (value == 5 && matrix[row * 5 + col] == -1) {
                values[5 * row + col] = 1;
            }
        }
    }
}

__global__ void solve(int *grids, int *constraints, int *constraint_sizes, int gridCount) {
    int futoshiki[25]; // Each thread will have a local copy
    int constraintSizeForPuzzle;

    int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalId < gridCount) {
        constraintSizeForPuzzle = constraint_sizes[globalId];

        int localConstraints[60];

        // Create local copy for puzzle (OK)
        for (int i = 0; i < 25; i++) {
            futoshiki[i] = grids[globalId * 25 + i];
        }

        int constraintStartIndex = 0;
        for (int i = 0; i < globalId; i++) {
            constraintStartIndex += constraint_sizes[i] * 4;
        }

        // Create local copy for constraints (OK)
        for (int i = 0; i < constraintSizeForPuzzle; i++) {
            localConstraints[4 * i] = constraints[constraintStartIndex + 4 * i];
            localConstraints[4 * i + 1] = constraints[constraintStartIndex + 4 * i + 1];
            localConstraints[4 * i + 2] = constraints[constraintStartIndex + 4 * i + 2];
            localConstraints[4 * i + 3] = constraints[constraintStartIndex + 4 * i + 3];
        }

        solveSingleThread(futoshiki, localConstraints, constraint_sizes[globalId]);

        /*if (!isSolutionCorrect(futoshiki, localConstraints, constraint_sizes[globalId]))
            printf("Solution is not correct! Thread ID: %d \n", globalId);*/

        // Write back to solutions vector
        for (int i = 0; i < 25; i++) {
            grids[globalId * 25 + i] = futoshiki[i];
        }

        /* if (globalId == 0) {
            printf("********************* Solution ********************* \n");
            for (int i = 0; i < 25; i += 5) {
                printf(" %d ", grids[25 * globalId + i]);
                printf(" %d ", grids[25 * globalId + i + 1]);
                printf(" %d ", grids[25 * globalId + i + 2]);
                printf(" %d ", grids[25 * globalId + i + 3]);
                printf(" %d ", grids[25 * globalId + i + 4]);
                printf("\n");
            }
            printf("********************* Solution ********************* \n");
        } */
    }
}



//You can change any part in order to optimize.
//Just don't forget to measure times

int main(int argc, char **argv) {

    std::string filename(argv[1]);
    std::ifstream file(filename.c_str());
    std::ifstream scout(filename.c_str());

    int no_grids;
    file >> no_grids;

    int dummy;
    scout >> dummy;

    int ***grids = new int **[no_grids];
    int **constraints = new int *[no_grids];

    for (int i = 0; i < no_grids; i++) {
        grids[i] = new int *[SIZE];
        for (int j = 0; j < SIZE; j++) {
            grids[i][j] = new int[SIZE];
        }
    }

    int elem0, elem1, elem2, elem3, elem4;
    int pre_cursor = 0;
    int cursor = 0;
    int csize = 0;

    std::string file_line;
    std::string scout_line;

    int *constraint_sizes = new int[no_grids];

    std::getline(scout, scout_line);//These are for spare lines
    std::getline(scout, scout_line);
    for (int i = 0; i < INPUTSIZE; i++) {
        std::getline(scout, scout_line);
        if (scout_line == "-------") {
            csize = i - pre_cursor - 5;
            constraint_sizes[cursor] = csize;
            cursor++;
            pre_cursor = i + 1;
        }
    }

    for (int i = 0; i < no_grids; i++) {
        constraints[i] = new int[constraint_sizes[i] * 4];
    }

    int NUM_CONSTRAINTS_ELEMENTS = 0;
    std::getline(file, file_line);
    for (int i = 0; i < no_grids; i++) {
        std::getline(file, file_line);
        for (int j = 0; j < SIZE; j++) {
            std::getline(file, file_line);
            //std::cout << "i: " << i << " file_line: " << file_line << std::endl;
            std::istringstream iss(file_line);
            iss >> elem0 >> elem1 >> elem2 >> elem3 >> elem4;
            grids[i][j][0] = elem0;
            grids[i][j][1] = elem1;
            grids[i][j][2] = elem2;
            grids[i][j][3] = elem3;
            grids[i][j][4] = elem4;
        }
        for (int c = 0; c < constraint_sizes[i]; c++) {
            std::getline(file, file_line);
            //std::cout << "i: " << i << "c line: " << file_line << std::endl;
            std::istringstream iss(file_line);
            iss >> elem0 >> elem1 >> elem2 >> elem3;
            constraints[i][4 * c] = elem0;
            constraints[i][4 * c + 1] = elem1;
            constraints[i][4 * c + 2] = elem2;
            constraints[i][4 * c + 3] = elem3;

            NUM_CONSTRAINTS_ELEMENTS += 4;
        }
    }

    // Flatten 3D Grids
    int *grids_1d = new int[no_grids * SIZE * SIZE];
    for (int numGrid = 0; numGrid < no_grids; numGrid++) {
        for (int row = 0; row < SIZE; row++) {
            for (int col = 0; col < SIZE; col++) {
                grids_1d[SIZE * SIZE * numGrid + SIZE * row + col] = grids[numGrid][row][col];
            }
        }
    }

    // Flatten 2D Constraints
    int counter = 0;
    int *constraints_1d = new int[NUM_CONSTRAINTS_ELEMENTS];
    for (int numGrid = 0; numGrid < no_grids; numGrid++) {
        int totalIterationCount = constraint_sizes[numGrid] * 4;
        for (int c = 0; c < totalIterationCount; c++) {
            constraints_1d[counter] = constraints[numGrid][c];
            counter++;
        }
    }

    /* You can access input and constraints like this
    for(int in = 0; in < 25; in++){
      std::cout << "in: " << in << std::endl;
      for(int i = 0; i < SIZE; i++){
        for(int j = 0; j < SIZE; j++){
      std::cout << grids[in][i][j] << " ";
        }
        std::cout << std::endl;
      }

      for(int i = 0; i < constraint_sizes[in]; i++){
        std::cout << constraints[in][4*i] << " " << constraints[in][4*i+1] << " " << constraints[in][4*i+2] << " " << constraints[in][4*i+3] << std::endl;
      }

    }
    */

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    //YOUR MEMORY OPERATIONS//Time accordingly
    int *d_grids;
    int *d_constraints;
    int *d_constraint_sizes;

    hipEventRecord(start, 0);
    std::cout << "GPU Memory Allocation" << std::endl;
    hipMalloc((void **) &d_grids, sizeof(int) * no_grids * SIZE * SIZE);
    hipMalloc((void **) &d_constraints, sizeof(int) * NUM_CONSTRAINTS_ELEMENTS);
    hipMalloc((void **) &d_constraint_sizes, sizeof(int) * no_grids);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GPU Memory allocation duration: %f ms \n", time);


    hipEventRecord(start, 0);
    std::cout << "CPU to GPU" << std::endl;
    hipMemcpy(d_grids, grids_1d, sizeof(int) * no_grids * SIZE * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_constraints, constraints_1d, sizeof(int) * NUM_CONSTRAINTS_ELEMENTS, hipMemcpyHostToDevice);
    hipMemcpy(d_constraint_sizes, constraint_sizes, sizeof(int) * no_grids, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("CPU to GPU Data Transfer Duration: %f ms \n", time);

//YOUR MEMORY OPERATIONS//


//KERNEL CALL//Time accordingly

    int threadPerBlock = 64;
    int blockCount = (no_grids / threadPerBlock) + 1;

//KERNEL CALL//
    hipEventRecord(start, 0);
    solve << < blockCount, threadPerBlock >> > (d_grids, d_constraints, d_constraint_sizes, no_grids);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Kernel Duration: %f ms \n", time);

    //YOUR MEMORY OPERARIONS//Time accordingly
    hipEventRecord(start, 0);
    std::cout << "GPU to CPU" << std::endl;
    hipMemcpy(grids_1d, d_grids, sizeof(int) * no_grids * SIZE * SIZE, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GPU to CPU Data Transfer Duration: %f ms \n", time);

    /*int gridNum = 122310;
    std::cout << "THESE ARE RESULTS: *********************" << std::endl;
    for (int i = 25 * gridNum; i < 25 * gridNum + 25; i += 5) {
        std::cout << grids_1d[i] << " ";
        std::cout << grids_1d[i + 1] << " ";
        std::cout << grids_1d[i + 2] << " ";
        std::cout << grids_1d[i + 3] << " ";
        std::cout << grids_1d[i + 4] << " ";
        std::cout << std::endl;
    }
    std::cout << "RESULTS ENDED *********************" << std::endl;*/


//YOUR MEMORY OPERARIONS//
    hipEventRecord(start, 0);
    std::cout << "Memory Deallocation" << std::endl;
    hipFree(d_grids);
    hipFree(d_constraints);
    hipFree(d_constraint_sizes);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Memory Deallocation Duration: %f ms \n", time);


    // Create and open a text file
    std::ofstream MyFile("_solution.txt");

    // Write to file
    MyFile << no_grids << std::endl;
    for (int i = 0; i < no_grids; i++) {
        MyFile << "-------" << std::endl;

        for (int j = 0; j < 25; j += 5) {
            MyFile << grids_1d[25 * i + j];
            MyFile << " " << grids_1d[25 * i + j + 1];
            MyFile << " " << grids_1d[25 * i + j + 2];
            MyFile << " " << grids_1d[25 * i + j + 3];
            MyFile << " " << grids_1d[25 * i + j + 4];
            MyFile << std::endl;
        }
    }
    MyFile << "-------" << std::endl;

    // Close the file
    MyFile.close();


    //Deallocate
    for (int i = 0; i < no_grids; i++) {
        for (int j = 0; j < SIZE; j++) {
            delete[] grids[i][j];
        }
        delete[] grids[i];
    }
    delete[] grids;

    for (int i = 0; i < no_grids; i++) {
        delete[] constraints[i];
    }
    delete[] constraints;

    delete[] constraint_sizes;

    delete[] grids_1d;
}